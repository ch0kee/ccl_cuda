#include "hip/hip_runtime.h"
/* Connected-component labeling */
/*

1) load (obj) scene
2) voxelize
3) display voxel grid
4) connected-component labeling
*/
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <memory>
#include <tuple>
#include <vector>

#define LOG(x) std::cout << __FUNCTION__ << ": "

class Grid {
public:
    void allocate(size_t x_size, size_t y_size, size_t z_size) {
        m_dimensions = std::make_tuple(x_size, y_size, z_size);

        m_cell_count = (x_size * y_size * z_size);
        size_t required_bytes = m_cell_count;// / 8;
        LOG(INFO) << "required bytes: " << required_bytes << std::endl;

        m_voxels.resize(required_bytes);
        std::fill(m_voxels.begin(), m_voxels.end(), false);
        std::fill(m_labels.begin(), m_labels.end(), 0); //0 means no label
    }

    auto x_size() const { return std::get<0>(m_dimensions); }
    auto y_size() const { return std::get<1>(m_dimensions); }
    auto z_size() const { return std::get<2>(m_dimensions); }

    bool is_black(size_t x, size_t y, size_t z) const {
        return m_voxels[index(x, y, z)];
    }

    bool is_white(size_t x, size_t y, size_t z) const {
        return !is_black(x, y, z);
    }

    void set_black(size_t x, size_t y, size_t z) {
        m_voxels[index(x, y, z)] = true;
    }

    void set_white(size_t x, size_t y, size_t z) {
        m_voxels[index(x, y, z)] = false;
    }

    size_t index(size_t x, size_t y, size_t z) const {
        return x + y * (x_size()) + z * (x_size() * y_size());
    }

    size_t cell_count() const {
        return m_cell_count;
    }

    size_t get_label(size_t x, size_t y, size_t z) const {
        return m_labels[index(x, y, z)];
    }

    void set_label(size_t l, size_t x, size_t y, size_t z) {
        m_labels[index(x, y, z)] = l;
    }

    bool has_label(size_t x, size_t y, size_t z) const {
        return m_labels[index(x, y, z)] != 0;
    }



private:


    std::vector<bool> m_voxels;
    std::vector<size_t> m_labels;
    
    std::tuple<size_t, size_t, size_t> m_dimensions;
    size_t m_cell_count{ 0 };


};

void ccl_cpu_two_phase(Grid& grid) {
    //Rosenfeld and Pfaltz 
    using label_t = size_t;


    std::vector<label_t> current_labels;
    std::vector<label_t> minimal_equivalent_label;

    //first phase
    for (size_t i = 0; i < grid.x_size(); ++i) {
        for (size_t j = 0; j < grid.y_size(); ++j) {
            for (size_t k = 0; k < grid.z_size(); ++k) {
                if (grid.is_black(i, j, k)) {
                    //if no neighbor has a label, assign a new label
                    


                }

            }
        }
    }

}

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    Grid g;
    g.allocate(100, 100, 100);
    label_cpu_two_phase(g);

    return 0;
}
